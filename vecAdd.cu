
#include <hip/hip_runtime.h>
__global__ void vecAddKernel(float *C, float *A, float *B, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    C[i] = A[i] + B[i];
  }
}

int main() {
  int size = 1 << 20;
  float *C, *A, *B;
  hipMallocManaged(&C, sizeof(float) * size);
  hipMallocManaged(&A, sizeof(float) * size);
  hipMallocManaged(&B, sizeof(float) * size);
}